#include "kernel_config.cuh"

namespace traccc::cuda::kernels {
__device__ finding_config g_finding_cfg;

void load_finding_config(const finding_config& cfg) {
    hipMemcpyToSymbol(HIP_SYMBOL(g_finding_cfg), &cfg, sizeof(finding_config));
}
}  // namespace traccc::cuda::kernels
